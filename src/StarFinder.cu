#include "StarFinder.h"

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/count.h>
#include <thrust/transform.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/zip_iterator.h>

#include <opencv2/cudaarithm.hpp>

#include <stdint.h>
#include <iostream>

using Byte = uint8_t;

struct IsNonzero
{
	IsNonzero() {}

	__host__ __device__
		bool operator()( const thrust::tuple<Byte, int> tup )
	{
		return thrust::get<0>( tup ) != 0;
	}
};

struct PixelIdxToLocation
{
	int m_nStep;
	PixelIdxToLocation( int nStep ) : m_nStep( nStep ) {}

	__host__ __device__
	thrust::pair<int, int> operator()( const int idx )
	{
		int x = idx % m_nStep;
		int y = idx / m_nStep;
		return thrust::make_pair( x, y );
	}
};

std::vector<std::pair<int, int>> FindStarsInImage( cv::cuda::GpuMat& dBoolImg )
{
	if ( dBoolImg.type() != CV_8U || dBoolImg.empty() || dBoolImg.isContinuous() == false )
		throw std::runtime_error( "Error: Stars must be found in boolean images!" );
	
	// Construct a device vector that we can iterate over from the mat's data
	using BytePtr = thrust::device_ptr<Byte>;

	using IdxVec = thrust::device_vector<int>;
	IdxVec dvIndices;
	
	// Create iterator to gives us pixel index (1-D)
	using CountIter = thrust::counting_iterator<int>;
	CountIter itCountBegin( 0 );
	CountIter itCountEnd( dBoolImg.size().area() );

	// Create an iterator that zips the pixel values with their 1-D index
	using PixelAndIdxIter = thrust::zip_iterator <thrust::tuple<BytePtr, CountIter>>;
	PixelAndIdxIter itPixAndIdxBegin = thrust::make_zip_iterator( thrust::make_tuple( BytePtr( (Byte *)dBoolImg.datastart ), itCountBegin ) );
	PixelAndIdxIter itPixAndIdxEnd = thrust::make_zip_iterator( thrust::make_tuple( BytePtr( (Byte *) dBoolImg.dataend ), itCountEnd ) );

	// Count the number of non-zero pixels (we need this so we can appropriately size dest vector)
	size_t count = thrust::count_if( itPixAndIdxBegin, itPixAndIdxEnd, IsNonzero() );
	dvIndices.resize( count );

	// Copy 1-D indices for non-zero pixels
	thrust::copy_if( itPixAndIdxBegin, itPixAndIdxEnd, thrust::make_zip_iterator( thrust::make_tuple( thrust::discard_iterator<>(), dvIndices.begin() ) ), IsNonzero() );

	// Transform this range into 2D coordinates
	using CoordVec = thrust::device_vector<thrust::pair<int, int>>;
	CoordVec dvNonzerPixelLocations( count );
	thrust::transform( dvIndices.begin(), dvIndices.end(), dvNonzerPixelLocations.begin(), PixelIdxToLocation( dBoolImg.step ) );

	hipDeviceSynchronize();

	// Download to host and return
	std::vector < thrust::pair<int, int>> vRet( dvNonzerPixelLocations.size() );
	thrust::copy( dvNonzerPixelLocations.begin(), dvNonzerPixelLocations.end(), vRet.begin() );
	std::vector<std::pair<int, int>> vRet2;
	for ( thrust::pair<int, int>& p : vRet )
		vRet2.emplace_back( p.first, p.second );
	return vRet2;
}