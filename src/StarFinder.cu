#include "StarFinder.h"

// I doubt I'm using all of these...
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/count.h>
#include <thrust/transform.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/zip_iterator.h>

#include <opencv2/cudaarithm.hpp>

#include <stdint.h>
#include <iostream>

using Byte = uint8_t;

// Look at zipped pixel and index, determine if pixel is nonzero
struct IsNonzero
{
	IsNonzero() {}

	__host__ __device__
		bool operator()( const thrust::tuple<Byte, int> tup )
	{
		return thrust::get<0>( tup ) != 0;
	}
};

// Convert 1-d pixel to star circle with radius
struct PixelToStarCircle
{
	// We need the image step to compute this
	float m_fRadius;
	int m_nStep;
	PixelToStarCircle( float fRadius, int nStep ) : m_fRadius( fRadius ), m_nStep( nStep ) {}

	__host__ __device__
	Circle operator()( const int idx )
	{
		int x = idx % m_nStep;
		int y = idx / m_nStep;
		return { (float) x, (float) y, m_fRadius };
	}
};

// Find non-zero pixel locations and return a vector of their pixel coordinates
std::vector<Circle> FindStarsInImage( float fStarRadius, cv::cuda::GpuMat& dBoolImg )
{
	// We need a contiguous image of bytes (which we'll be treating as bools)
	if ( dBoolImg.type() != CV_8U || dBoolImg.empty() || dBoolImg.isContinuous() == false )
		throw std::runtime_error( "Error: Stars must be found in boolean images!" );

	// Construct a device vector that we can iterate over from the mat's data
	using BytePtr = thrust::device_ptr<Byte>;

	// Create iterator to gives us pixel index (1-D)
	using CountIter = thrust::counting_iterator<int>;
	CountIter itCountBegin( 0 );
	CountIter itCountEnd( dBoolImg.size().area() );

	// Create an iterator that zips the pixel values with their 1-D index
	using PixelAndIdxIter = thrust::zip_iterator <thrust::tuple<BytePtr, CountIter>>;
	PixelAndIdxIter itPixAndIdxBegin = thrust::make_zip_iterator( thrust::make_tuple( BytePtr( (Byte *)dBoolImg.datastart ), itCountBegin ) );
	PixelAndIdxIter itPixAndIdxEnd = thrust::make_zip_iterator( thrust::make_tuple( BytePtr( (Byte *) dBoolImg.dataend ), itCountEnd ) );

	// Count the number of non-zero pixels (we need this so we can appropriately size dest vector)
	using IdxVec = thrust::device_vector<int>;
	IdxVec dvIndices;
	size_t count = thrust::count_if( itPixAndIdxBegin, itPixAndIdxEnd, IsNonzero() );
	dvIndices.resize( count );

	// Copy 1-D indices for non-zero pixels, discard the pixel values
	thrust::copy_if( itPixAndIdxBegin, itPixAndIdxEnd, thrust::make_zip_iterator( thrust::make_tuple( thrust::discard_iterator<>(), dvIndices.begin() ) ), IsNonzero() );

	// Transform this range into star circles
	using CircleVec = thrust::device_vector<Circle>;
	CircleVec dvStarCircles( count );
	thrust::transform( dvIndices.begin(), dvIndices.end(), dvStarCircles.begin(), PixelToStarCircle( fStarRadius, dBoolImg.step ) );

	// Don't know if this is necessary
	hipDeviceSynchronize();

	// Download to host
	std::vector<Circle> hvStarCircles( dvStarCircles.size() );
	thrust::copy( dvStarCircles.begin(), dvStarCircles.end(), hvStarCircles.begin() );

	// Collapse
	std::vector<Circle> vStarPos_Collapsed = CollapseCircles( hvStarCircles );

	// Return collapsed star positions
	return vStarPos_Collapsed;
}
