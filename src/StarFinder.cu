#include "StarFinder.h"

// I doubt I'm using all of these...
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/count.h>
#include <thrust/transform.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/zip_iterator.h>

#include <opencv2/cudaarithm.hpp>

#include <stdint.h>
#include <iostream>

using Byte = uint8_t;

// Look at zipped pixel and index, determine if pixel is nonzero
struct IsNonzero
{
	IsNonzero() {}

	__host__ __device__
		bool operator()( const thrust::tuple<Byte, int> tup )
	{
		return thrust::get<0>( tup ) != 0;
	}
};

// Convert 1D pixel index to 2D pixel coordinate
struct PixelIdxToLocation
{
	// We need the image step to compute this
	int m_nStep;
	PixelIdxToLocation( int nStep ) : m_nStep( nStep ) {}

	__host__ __device__
	thrust::pair<int, int> operator()( const int idx )
	{
		int x = idx % m_nStep;
		int y = idx / m_nStep;
		return thrust::make_pair( x, y );
	}
};

// Find non-zero pixel locations and return a vector of their pixel coordinates
std::vector<std::pair<int, int>> FindStarsInImage( cv::cuda::GpuMat& dBoolImg )
{
	// We need a contiguous image of bytes (which we'll be treating as bools)
	if ( dBoolImg.type() != CV_8U || dBoolImg.empty() || dBoolImg.isContinuous() == false )
		throw std::runtime_error( "Error: Stars must be found in boolean images!" );

	// Construct a device vector that we can iterate over from the mat's data
	using BytePtr = thrust::device_ptr<Byte>;

	// Create iterator to gives us pixel index (1-D)
	using CountIter = thrust::counting_iterator<int>;
	CountIter itCountBegin( 0 );
	CountIter itCountEnd( dBoolImg.size().area() );

	// Create an iterator that zips the pixel values with their 1-D index
	using PixelAndIdxIter = thrust::zip_iterator <thrust::tuple<BytePtr, CountIter>>;
	PixelAndIdxIter itPixAndIdxBegin = thrust::make_zip_iterator( thrust::make_tuple( BytePtr( (Byte *)dBoolImg.datastart ), itCountBegin ) );
	PixelAndIdxIter itPixAndIdxEnd = thrust::make_zip_iterator( thrust::make_tuple( BytePtr( (Byte *) dBoolImg.dataend ), itCountEnd ) );

	// Count the number of non-zero pixels (we need this so we can appropriately size dest vector)
	using IdxVec = thrust::device_vector<int>;
	IdxVec dvIndices;
	size_t count = thrust::count_if( itPixAndIdxBegin, itPixAndIdxEnd, IsNonzero() );
	dvIndices.resize( count );

	// Copy 1-D indices for non-zero pixels, discard the pixel values
	thrust::copy_if( itPixAndIdxBegin, itPixAndIdxEnd, thrust::make_zip_iterator( thrust::make_tuple( thrust::discard_iterator<>(), dvIndices.begin() ) ), IsNonzero() );

	// Transform this range into 2D coordinates
	using CoordVec = thrust::device_vector<thrust::pair<int, int>>;
	CoordVec dvNonzerPixelLocations( count );
	thrust::transform( dvIndices.begin(), dvIndices.end(), dvNonzerPixelLocations.begin(), PixelIdxToLocation( dBoolImg.step ) );

	// Don't know if this is necessary
	hipDeviceSynchronize();

	// Download to host and return (need to create a better "pair" object)
	std::vector < thrust::pair<int, int>> hvNonzerPixelLocations( dvNonzerPixelLocations.size() );
	thrust::copy( dvNonzerPixelLocations.begin(), dvNonzerPixelLocations.end(), hvNonzerPixelLocations.begin() );
	std::vector<std::pair<int, int>> vRet;
	for ( thrust::pair<int, int>& p : hvNonzerPixelLocations )
		vRet.emplace_back( p.first, p.second );
	return vRet;
}
